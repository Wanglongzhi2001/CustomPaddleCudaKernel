#include "hip/hip_runtime.h"
#include "utils.h"
#include "stdio.h"
template <unsigned int blockSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (blockSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (blockSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (blockSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (blockSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (blockSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}


#define WARP_SIZE 32

template <int blockSize>
__device__ float blockReduceSum(float sum){
    // Shared mem for partial sums (one per warp in the block)
    static __shared__ float warpLevelSums[WARP_SIZE]; 
    const int laneId = threadIdx.x % WARP_SIZE;
    const int warpId = threadIdx.x / WARP_SIZE;

    sum = warpReduceSum<blockSize>(sum);

    if(laneId == 0 )warpLevelSums[warpId] = sum;
    __syncthreads();
    // read from shared memory only if that warp existed
    sum = (threadIdx.x < blockDim.x / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    // Final reduce using first warp
    if (warpId == 0) sum = warpReduceSum<blockSize/WARP_SIZE>(sum); 
    // write result for this block to global mem
    return sum;
}

template<int blockSize>
__global__ void rms_norm_kernel(
    const float* __restrict__ input,
    const float* __restrict__ norm_weight,
    float* __restrict__ output,
    const float* epsilon,
    const int num_tokens,
    const int hidden_size) {
    
    // 每个 block 的 variance
    __shared__ float s_variance;

    const int tid = threadIdx.x;
    if (tid == 0) printf("you have got 3\n");
    // 平方值，blockReduce 统计
    float variance = 0.0f;

    // 先用网格跨步循环将元素 offload 到一个 block 内的元素
    for (int i = tid; i < hidden_size; i += blockDim.x) {
        const int input_idx = blockIdx.x * hidden_size + i;
        variance += input[input_idx] * input[input_idx];
    }

    if (tid == 0) printf("you have got 4\n");
    // block ReduceSum 
    variance = blockReduceSum<blockSize>(variance);
    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / hidden_size + epsilon[0]);
    }
    if (tid == 0) printf("you have got 5\n");

    __syncthreads();
    for (int i = tid; i < hidden_size; i += blockDim.x) {
        const int input_idx = blockIdx.x * hidden_size + i;
        const int output_idx = blockIdx.x * hidden_size + i;
        output[output_idx] = input[input_idx] * s_variance * norm_weight[i];
    }
}

void launchRMSNorm(paddle::Tensor& input, paddle::Tensor& weight, paddle::Tensor& epsilon, paddle::Tensor& output) {
    printf("you have got 1\n");
    int hidden_size = input.dims()[1];
    int num_tokens = input.dims()[0];
    const int blockSize = 1024;
    dim3 grid(num_tokens);
    dim3 block(blockSize);
    auto stream = input.stream();

    printf("you have got 2\n");
    // 一个 block 计算一行数据
    rms_norm_kernel<blockSize><<<grid, block, 0, stream>>>(input.data<float>(), weight.data<float>(), output.data<float>(), epsilon.data<float>(), num_tokens, hidden_size);
}



void MyRMSNorm(paddle::Tensor& input,
            paddle::Tensor& weight,
            paddle::Tensor&  epsilon,
            paddle::Tensor& output) {
    launchRMSNorm(input, weight, epsilon, output);
}

PD_BUILD_OP(my_rms_norm)
    .Inputs({"input", "weight", "epsilon", "output"})
    .Outputs({"Out"})
    .SetInplaceMap({{"output", "Out"}})
    .SetKernelFn(PD_KERNEL(MyRMSNorm));
