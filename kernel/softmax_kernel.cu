#include "hip/hip_runtime.h"
#include "utils.h"

#define WARP_SIZE 32

__device__ __forceinline__ float warpReduceSum(float sum, int blockSize) {
    if (blockSize >= 32) sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (blockSize >= 16) sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (blockSize >= 8) sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (blockSize >= 4) sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (blockSize >= 2) sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

__device__ __forceinline__ float warpReduceMax(float max, int blockSize) {
    if (blockSize >= 32) max = fmaxf(max, __shfl_down_sync(0xffffffff, max, 16)); // 0-16, 1-17, 2-18, etc.
    if (blockSize >= 16) max = fmaxf(max, __shfl_down_sync(0xffffffff, max, 8));// 0-8, 1-9, 2-10, etc.
    if (blockSize >= 8) max = fmaxf(max, __shfl_down_sync(0xffffffff, max, 4));// 0-4, 1-5, 2-6, etc.
    if (blockSize >= 4) max = fmaxf(max, __shfl_down_sync(0xffffffff, max, 2));// 0-2, 1-3, 4-6, 5-7, etc.
    if (blockSize >= 2) max = fmaxf(max, __shfl_down_sync(0xffffffff, max, 1));// 0-1, 2-3, 4-5, etc.
    return max;
}


// 一个 block 计算一行的数据，一个线程处理一个数据
__global__ void softmax_v1(
                        const float* __restrict__ input,
                        float* __restrict__ output,
                        const int total_elem_cnt,
                        float* __restrict__ block_sum,
                        float* __restrict__ block_max,
                        int blockSize) {
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;

    float max_val = input[idx];
    float max = warpReduceMax(max_val, blockSize);
    // 这里设置成 128 就代表着 block 数目不能超过 128
    extern __shared__ float smem[];
    const int lane_id = tid % WARP_SIZE;
    const int warp_id = tid / WARP_SIZE;

    if(lane_id == 0) smem[warp_id] = max;
    __syncthreads();
    max = (threadIdx.x < blockSize / WARP_SIZE) ? smem[lane_id] : 0;
    if (warp_id == 0) max = warpReduceMax(max, blockSize/WARP_SIZE); 
    if (tid == 0) block_max[blockIdx.x] = max;
    __syncthreads();
    // 数值稳定
    float exp_val = expf(input[idx] - block_max[blockIdx.x]);

    // 先做 warp 级别的 blockReduce
    float sum = warpReduceSum(exp_val, blockSize);
    // 再使用 total 汇总所有 warp 的 sum 得到一行总的 sum


    if(lane_id == 0) smem[warp_id] = sum;
    __syncthreads();
    sum = (threadIdx.x < blockSize / WARP_SIZE) ? smem[lane_id] : 0;
    if (warp_id == 0) sum = warpReduceSum(sum, blockSize/WARP_SIZE); 
    if (tid == 0) block_sum[blockIdx.x] = sum;
    __syncthreads();
    if (idx < total_elem_cnt) output[idx] = exp_val / block_sum[blockIdx.x]; 
}

void launchSoftmaxV1(paddle::Tensor& x, paddle::Tensor& output) {
    int M = x.dims()[0];
    int N = x.dims()[1];
    int elem_cnt = x.numel();
    const int BLOCK_SIZE = N;
    int block_num = M;
    dim3 Grid( block_num, 1);
    dim3 Block( BLOCK_SIZE, 1);
    auto stream = x.stream();
    float *block_sum;
    float *block_max;
    hipMalloc((void **)&block_sum, M * sizeof(float));
    hipMalloc((void **)&block_max, M * sizeof(float));
    const int smem_size = M;
    softmax_v1<<<Grid, Block, smem_size>>>(x.data<float>(), output.data<float>(), elem_cnt, block_sum, block_max, BLOCK_SIZE);
}


std::vector<paddle::Tensor> MySoftmax(paddle::Tensor& x) {
    auto output = paddle::full(x.shape(), 0, x.dtype(), x.place());
    launchSoftmaxV1(x, output);
    return {output};
}

std::vector<std::vector<int64_t>> MySoftmaxInferShape(const std::vector<int64_t>& x_shape) {
    return {x_shape};
}

std::vector<paddle::DataType> MySoftmaxInferDtype(const paddle::DataType& x_dtype) {
    return {x_dtype};
}


PD_BUILD_OP(my_softmax)
    .Inputs({"x"})
    .Outputs({"out"})
    .SetKernelFn(PD_KERNEL(MySoftmax))
    .SetInferShapeFn(PD_INFER_SHAPE(MySoftmaxInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(MySoftmaxInferDtype));
